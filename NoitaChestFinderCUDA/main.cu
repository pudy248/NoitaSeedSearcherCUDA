#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include "misc/error.h"

#include "Precheckers.h"
#include "Worldgen.h"
#include "WorldgenSearch.h"
#include "Filters.h"

#include <iostream>
#include <fstream>
#include <chrono>
#include <thread>

//#define SEED_OUTPUT

//tired of seeing an error for it being undefined
__device__ int atomicAdd(int* address, int val);

struct GlobalConfig
{
	uint startSeed;
	uint endSeed;
	int printInterval;
};

struct MemBlockSizes
{
	size_t outputSize;
	size_t mapDataSize;
	size_t miscMemSize;
	size_t visitedMemSize;
};

__global__ void Kernel(byte* outputBlock, byte* dMapData, byte* dMiscMem, byte* dVisitedMem, MemBlockSizes memSizes, GlobalConfig globalCfg, PrecheckConfig precheckCfg, WorldgenConfig worldCfg, LootConfig lootCfg, FilterConfig filterCfg, int* checkedSeeds, int* passedSeeds)
{
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	uint stride = blockDim.x * gridDim.x;
	for (int seed = globalCfg.startSeed + index; seed < globalCfg.endSeed; seed += stride)
	{
#ifdef SEED_OUTPUT
		byte* output = outputBlock + (seed - globalCfg.startSeed) * memSizes.outputSize;
#else
		byte* output = outputBlock + index * memSizes.outputSize;
#endif
#ifdef DO_WORLDGEN
		byte* map = dMapData + index * memSizes.mapDataSize;
		byte* miscMem = dMiscMem + index * memSizes.miscMemSize;
		byte* visited = dVisitedMem + index * memSizes.visitedMemSize;
		byte* spawnableMem = miscMem;
#endif
		if (!PrecheckSeed(seed, precheckCfg))
		{
			atomicAdd(checkedSeeds, 1);
			continue;
		}

		bool seedPassed = true;

#ifdef DO_WORLDGEN
		GenerateMap(seed, output, map, visited, miscMem, worldCfg, globalCfg.startSeed / 5);

		CheckSpawnables(map, seed, spawnableMem, output, worldCfg, lootCfg, memSizes.miscMemSize);
		
		SpawnableBlock result = ParseSpawnableBlock(spawnableMem, map, output, lootCfg, memSizes.mapDataSize);
		seedPassed = SpawnablesPassed(result, filterCfg, true);
#endif

		atomicAdd(checkedSeeds, 1);
		if (seedPassed)
		{
			printf("Seed passed: %i\n", seed);
			atomicAdd(passedSeeds, 1);
		}
	}
}

/*
__global__ void wandExperiment(const int level, const bool nonShuffle)
{
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	uint stride = blockDim.x * gridDim.x;

	constexpr int radius = 100;
	constexpr int seed = 913380622;
	constexpr int center_x = 5061;
	constexpr int center_y = 11119;
	for (int x = -radius + index + center_x; x < radius + center_x; x += stride) {
		for (int y = -radius + center_y; y < radius + center_y; y++) {
			Wand w = GetWandWithLevel(seed, x, y, level, nonShuffle, false);
			bool found = false;
			for (int i = 0; i < w.spellIdx; i++)
			{
				if (w.spells[i] == SPELL_SWAPPER_PROJECTILE)
					found = true;
			}

			if (found) printf("%i %i\n", x, y);
		}
	}
}
*/

int main()
{
	/*
	hipSetDeviceFlags(hipDeviceMapHost);

	volatile int* h_buckets;
	volatile int* d_buckets;

	hipHostAlloc((void**)&h_buckets, sizeof(volatile int) * 70, hipHostMallocMapped);

	hipHostGetDevicePointer((void**)&d_buckets, (void*)h_buckets, 0);

	for (int i = 0; i < 70; i++) h_buckets[i] = 0;
	wandExperiment<<<256,64>>>((int*)d_buckets, 6, true);
	hipDeviceSynchronize();
	for (int i = 0; i < 70; i++) {
		printf("multicast %i: %i wands\n", i, h_buckets[i]);
	}
	return;*/

	for (int global_iters = 0; global_iters < 1; global_iters++)
	{
		chrono::steady_clock::time_point time1 = chrono::steady_clock::now();

		//MINES
		WorldgenConfig worldCfg = { 348, 448, 256, 103, 34, 14, true, false, 100 };
		const char* fileName = "minesWang.bin";
		constexpr auto NUMBLOCKS = 128;
		constexpr auto BLOCKSIZE = 64;
		constexpr auto biomeIdx = 0;
		constexpr auto mapMemMult = 4;
		constexpr auto miscMemMult = 10;

		//EXCAVATION SITE
		//WorldgenConfig worldCfg = { 344, 440, 409, 102, 31, 17, false, false, 100 };
		//const char* fileName = "excavationsiteWang.bin";
		//constexpr auto NUMBLOCKS = 64;
		//constexpr auto BLOCKSIZE = 64;
		//constexpr auto biomeIdx = 1;
		//constexpr auto mapMemMult = 4;
		//constexpr auto miscMemMult = 10;

		//SNOWCAVE
		//WorldgenConfig worldCfg = { 440, 560, 512, 153, 30, 20, false, false, 100 };
		//const char* fileName = "snowcaveWang.bin";
		//constexpr auto NUMBLOCKS = 64;
		//constexpr auto BLOCKSIZE = 32;
		//constexpr auto biomeIdx = 1;
		//constexpr auto mapMemMult = 4;
		//constexpr auto miscMemMult = 10;

		//CRYPT
		//WorldgenConfig worldCfg = { 282, 342, 717, 204, 26, 35, false, false, 100 };
		//const char* fileName = "cryptWang.bin";
		//constexpr auto NUMBLOCKS = 32;
		//constexpr auto BLOCKSIZE = 32;
		//constexpr auto biomeIdx = 10;
		//constexpr auto mapMemMult = 4;
		//constexpr auto miscMemMult = 10;

		//OVERGROWN CAVERNS
		//WorldgenConfig worldCfg = { 144, 235, 359, 461, 59, 16, false, false, 1 };
		//const char* fileName = "fungiforestWang.bin";
		//constexpr auto NUMBLOCKS = 32;
		//constexpr auto BLOCKSIZE = 32;
		//constexpr auto biomeIdx = 15;
		//constexpr auto mapMemMult = 4;
		//constexpr auto miscMemMult = 10;

		//HELL
		//WorldgenConfig worldCfg = { 156, 364, 921, 256, 25, 43, false, true, 100 };
		//const char* fileName = "hellWang.bin";
		//constexpr auto NUMBLOCKS = 32;
		//constexpr auto BLOCKSIZE = 32;
		//constexpr auto biomeIdx = 0;
		//constexpr auto mapMemMult = 4;
		//constexpr auto miscMemMult = 10;

		MemBlockSizes memSizes = {
#ifdef SEED_OUTPUT
			3 * worldCfg.map_w * worldCfg.map_h,
#else
			256,
#endif
			mapMemMult * 3 * worldCfg.map_w * (worldCfg.map_h + 4),
			miscMemMult * worldCfg.map_w * worldCfg.map_h,
			worldCfg.map_w * worldCfg.map_h
		};

		GlobalConfig globalCfg = { 1, INT_MAX, 5 };

		Item iF1[FILTER_OR_COUNT] = { PAHA_SILMA };
		Item iF2[FILTER_OR_COUNT] = { MIMIC };
		Material mF1[FILTER_OR_COUNT] = { BRASS };
		Spell sF1[FILTER_OR_COUNT] = { SPELL_OMEGA };
		Spell sF2[FILTER_OR_COUNT] = { SPELL_BLACK_HOLE_DEATH_TRIGGER, SPELL_BLACK_HOLE };
		//Spell sF3[FILTER_OR_COUNT] = { SPELL_BLACK_HOLE };

		ItemFilter iFilters[] = { ItemFilter(iF1, 4), ItemFilter(iF2) };
		MaterialFilter mFilters[] = { MaterialFilter(mF1) };
		SpellFilter sFilters[] = { SpellFilter(sF1), SpellFilter(sF2) };

		FilterConfig filterCfg = FilterConfig(true, 1, iFilters, 0, mFilters, 0, sFilters, false, 36);
		LootConfig lootCfg = LootConfig(0, 0, true, false, false, false, false, filterCfg.materialFilterCount > 0, false, biomeIdx, false);

		PrecheckConfig precheckCfg = {
			false,
			false, WATER,
			false, WATER,
			false, AlchemyOrdering::ONLY_CONSUMED, {MUD, WATER, SOIL}, {MUD, WATER, SOIL},
			false, {FungalShift(SS_ACID_GAS, SD_NONE, 0, 1)},
			false, {BM_GOLD_VEIN_SUPER, BM_NONE, BM_NONE},
			false, {{PERK_PERKS_LOTTERY, true, 0, 3}, {PERK_UNLIMITED_SPELLS, false, 0, 6}, {PERK_EDIT_WANDS_EVERYWHERE, false, 0, 3}, {PERK_PROTECTION_EXPLOSION, false, 0, 6}, {PERK_NO_MORE_SHUFFLE, false, 0, 6}},
			false, filterCfg, lootCfg,
			false, true, 0, 6
		};

		if (precheckCfg.checkBiomeModifiers && !ValidateBiomeModifierConfig(precheckCfg))
		{
			printf("Impossible biome modifier set! Aborting...\n");
			return;
		}

		int sharedMemSize = 0;
#ifdef SEED_OUTPUT
		size_t outputSize = (globalCfg.endSeed - globalCfg.startSeed) * memSizes.outputSize;
#else
		size_t outputSize = NUMBLOCKS * BLOCKSIZE * memSizes.outputSize;
#endif
		size_t tileDataSize = 3 * worldCfg.tiles_w * worldCfg.tiles_h;
		size_t mapDataSize = NUMBLOCKS * BLOCKSIZE * memSizes.mapDataSize;
		size_t miscMemSize = NUMBLOCKS * BLOCKSIZE * memSizes.miscMemSize;
		size_t visitedMemSize = NUMBLOCKS * BLOCKSIZE * memSizes.visitedMemSize;

		byte* output = (byte*)malloc(outputSize);
		byte* tileData = (byte*)malloc(3 * worldCfg.tiles_w * worldCfg.tiles_h);
		std::ifstream source(fileName, std::ios_base::binary);
		source.read((char*)tileData, 3 * worldCfg.tiles_w * worldCfg.tiles_h);
		source.close();

		byte* dTileData;
		byte* dOutput;
		byte* dMapData;
		byte* dMiscMem;
		byte* dVisitedMem;

		printf("Memory Usage Statistics:\n");
		printf("Output: %ziMB  Map data: %ziMB\n", outputSize / 1000000, mapDataSize / 1000000);
		printf("Misc memory: %ziMB  Visited cells: %ziMB\n", miscMemSize / 1000000, visitedMemSize / 1000000);
		printf("Total memory: %ziMB\n",(tileDataSize + outputSize + mapDataSize + miscMemSize + visitedMemSize) / 1000000);

		hipSetDeviceFlags(hipDeviceMapHost);

		volatile uint* h_checkedSeeds, * h_passedSeeds;
		volatile uint* d_checkedSeeds, * d_passedSeeds;

		hipHostAlloc((void**)&h_checkedSeeds, sizeof(volatile uint), hipHostMallocMapped);
		hipHostAlloc((void**)&h_passedSeeds, sizeof(volatile uint), hipHostMallocMapped);

		hipHostGetDevicePointer((void**)&d_checkedSeeds, (void*)h_checkedSeeds, 0);
		hipHostGetDevicePointer((void**)&d_passedSeeds, (void*)h_passedSeeds, 0);

		*h_checkedSeeds = 0;
		*h_passedSeeds = 0;

		checkCudaErrors(hipMalloc(&dOutput, outputSize));
#ifdef DO_WORLDGEN
		checkCudaErrors(hipMalloc(&dTileData, tileDataSize));
		checkCudaErrors(hipMalloc(&dMapData, mapDataSize));
		checkCudaErrors(hipMalloc(&dMiscMem, miscMemSize));
		checkCudaErrors(hipMalloc(&dVisitedMem, visitedMemSize));

		checkCudaErrors(hipMemcpy(dTileData, tileData, 3 * worldCfg.tiles_w * worldCfg.tiles_h, hipMemcpyHostToDevice));
		buildTS << <1, 1 >> > (dTileData, worldCfg.tiles_w, worldCfg.tiles_h);
		checkCudaErrors(hipDeviceSynchronize());
#endif

		hipEvent_t _event;
		checkCudaErrors(hipEventCreateWithFlags(&_event, hipEventDisableTiming));
		Kernel << <NUMBLOCKS, BLOCKSIZE, sharedMemSize >> > (dOutput, dMapData, dMiscMem, dVisitedMem, memSizes, globalCfg, precheckCfg, worldCfg, lootCfg, filterCfg, (int*)d_checkedSeeds, (int*)d_passedSeeds);
		checkCudaErrors(hipEventRecord(_event));

		int intervals = 0;
		if (globalCfg.printInterval > 0)
		{
			while (hipEventQuery(_event) == hipErrorNotReady && (*h_checkedSeeds) < (globalCfg.endSeed - globalCfg.startSeed))
			{
				intervals++;
				float percentComplete = ((float)(*h_checkedSeeds) / (globalCfg.endSeed - globalCfg.startSeed));
				printf("Interval %i: %2.4f%% complete (%i seeds), found %i valid seeds.\n", intervals, percentComplete * 100, *h_checkedSeeds, *h_passedSeeds);
				this_thread::sleep_for(chrono::seconds(globalCfg.printInterval));
			}
		}
		checkCudaErrors(hipDeviceSynchronize());

		freeTS << <1, 1 >> > ();
		checkCudaErrors(hipDeviceSynchronize());

		checkCudaErrors(hipMemcpy(output, dOutput, outputSize, hipMemcpyDeviceToHost));

		checkCudaErrors(hipFree(dOutput));

#ifdef DO_WORLDGEN
		checkCudaErrors(hipFree(dTileData));
		checkCudaErrors(hipFree(dMapData));
		checkCudaErrors(hipFree(dMiscMem));
		checkCudaErrors(hipFree(dVisitedMem));
#endif

		free(tileData);
		chrono::steady_clock::time_point time2 = chrono::steady_clock::now();
		std::chrono::nanoseconds duration = time2 - time1;

		printf("Intervals elapsed: %i (%ims). Checked %i seeds, found %i valid seeds.\n", intervals, (int)(duration.count() / 1000000), *h_checkedSeeds, *h_passedSeeds);


		std::ofstream f = ofstream("output.bin", std::ios::binary);
		f.write((char*)output, outputSize);
		f.close();
		free(output);
	}
}