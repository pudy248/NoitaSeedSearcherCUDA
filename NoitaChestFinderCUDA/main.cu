#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include "misc/error.h"

#include "Precheckers.h"
#include "Worldgen.h"
#include "WorldgenSearch.h"

#include <iostream>
#include <fstream>
#include <chrono>

#define NUMBLOCKS 128
#define BLOCKSIZE 64

struct GlobalConfig {
	uint startSeed;
	uint endSeed;
};

struct MemBlockSizes {
	size_t outputSize;
	size_t mapDataSize;
	size_t miscMemSize;
	size_t visitedMemSize;
};

__global__ void Kernel(byte* outputBlock, byte* dMapData, byte* dMiscMem, byte* dVisitedMem, MemBlockSizes memSizes, GlobalConfig globalCfg, PrecheckConfig precheckCfg, WorldgenConfig worldCfg, LootConfig lootCfg)
{
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	uint stride = blockDim.x * gridDim.x;

	for (int seed = globalCfg.startSeed + index; seed < globalCfg.endSeed; seed += stride) {

		//byte* output = outputBlock + (seed - globalCfg.startSeed) * memSizes.outputSize;
		byte* output = outputBlock + index * memSizes.outputSize;
		byte* map = dMapData + index * memSizes.mapDataSize;
		byte* miscMem = dMiscMem + index * memSizes.miscMemSize;
		byte* visited = dVisitedMem + index * memSizes.visitedMemSize;

		if (!PrecheckSeed(seed, precheckCfg)) continue;

		GenerateMap(seed, output, map, visited, miscMem, worldCfg, globalCfg.startSeed / 5);

		byte* localPtr1 = miscMem;
		CheckSpawnables(map, seed, &localPtr1, output, worldCfg, lootCfg);

		byte* localPtr2 = miscMem;
		SeedSpawnables result = ParseSpawnableBlock(&localPtr2, output, lootCfg);
		for (int i = 0; i < result.count; i++) {
			Spawnable s = result.spawnables[i];
			bool printSpawnable = false;
			for (int j = 0; j < s.count; j++) {
				if(s.contents[j] == TRUE_ORB) {
					printSpawnable = true;
					//byte* ptr = (byte*)(s.contents) + j + 1;
					//Material m = readMaterial(&ptr);
					//if (m == MONSTER_POWDER_TEST)
				}
			}

			if (printSpawnable) {
				printf("%i @ (%i, %i) (#%i of %i): T%i, %i bytes: (", result.seed, s.x, s.y, i + 1, result.count, s.sType, s.count);
				for (int n = 0; n < s.count; n++) printf("%x ", s.contents[n]);
				printf("\b)\n");
			}
		}
		freeSeedSpawnables(result);

		if (seed % 10'000'000 == 0) printf("Seed %i\n", seed);
	}
}

int main() 
{
	chrono::steady_clock::time_point time1 = chrono::steady_clock::now();

	const int tiles_w = 348;
	const int tiles_h = 448;
	const int map_w = 256;
	const int map_h = 103;

	MemBlockSizes memSizes = {
		1024,
		3 * map_w * (map_h + 4),
		sizeof(IntPair) * map_w * map_h,
		map_w * map_h
	};

	GlobalConfig globalCfg = { 1, 100000 };
	PrecheckConfig precheckCfg = {
		false,
		false, MATERIAL_NONE,
		false, URINE,
		false, {MUD, WATER, SOIL}, {MUD, WATER, SOIL},
		false, {},
		false, {CONDUCTIVE, MODIFIER_NONE, CONDUCTIVE, MODIFIER_NONE, CONDUCTIVE, CONDUCTIVE},
		false, {PERKS_LOTTERY, GAMBLE, EDIT_WANDS_EVERYWHERE, PROTECTION_EXPLOSION, PROTECTION_MELEE } };
	WorldgenConfig worldCfg = { tiles_w, tiles_h, map_w, map_h, 34, 14, true, 5 };
	LootConfig lootCfg = { 0, true, false, false, false, false, false, false, false };

	size_t tileDataSize = 3 * tiles_w * tiles_h;
	//size_t outputSize = (globalCfg.endSeed - globalCfg.startSeed) * memSizes.outputSize;
	size_t outputSize = NUMBLOCKS * BLOCKSIZE * memSizes.outputSize;
	size_t mapDataSize = NUMBLOCKS * BLOCKSIZE * memSizes.mapDataSize;
	size_t miscMemSize = NUMBLOCKS * BLOCKSIZE * memSizes.miscMemSize;
	size_t visitedMemSize = NUMBLOCKS * BLOCKSIZE * memSizes.visitedMemSize;

	byte* output = (byte*)malloc(outputSize);
	byte* tileData = (byte*)malloc(3 * tiles_w * tiles_h);
	std::ifstream source("minesDump.bin", std::ios_base::binary);
	source.read((char*)tileData, 3 * tiles_w * tiles_h);
	source.close();

	byte* dTileData;
	byte* dOutput;
	byte* dMapData;
	byte* dMiscMem;
	byte* dVisitedMem;

	//printf("Memory Usage Statistics:\n");
	//printf("Output: %iMB  Map data: %iMB\n", outputSize / 1000000, mapDataSize / 1000000);
	//printf("Misc memory: %iMB  Visited cells: %iMB\n", miscMemSize / 1000000, visitedMemSize / 1000000);
	//printf("Total memory: %iMB\n",(tileDataSize + outputSize + mapDataSize + miscMemSize + visitedMemSize) / 1000000);

	checkCudaErrors(hipMalloc(&dTileData, tileDataSize));
	checkCudaErrors(hipMalloc(&dOutput, outputSize));
	checkCudaErrors(hipMalloc(&dMapData, mapDataSize));
	checkCudaErrors(hipMalloc(&dMiscMem, miscMemSize));
	checkCudaErrors(hipMalloc(&dVisitedMem, visitedMemSize));

	checkCudaErrors(hipMemcpy(dTileData, tileData, 3 * tiles_w * tiles_h, hipMemcpyHostToDevice));
	buildTS << <1, 1 >> > (dTileData, tiles_w, tiles_h);
	checkCudaErrors(hipDeviceSynchronize());
	int sharedMemSize = 0;
	//printf("kernel shared mem: %i\n", sharedMemSize);
	Kernel << <NUMBLOCKS, BLOCKSIZE, sharedMemSize >> > (dOutput, dMapData, dMiscMem, dVisitedMem, memSizes, globalCfg, precheckCfg, worldCfg, lootCfg);
	checkCudaErrors(hipDeviceSynchronize());
	//printf("exit kernel\n");
	freeTS << <1, 1 >> > ();
	checkCudaErrors(hipDeviceSynchronize());

	//checkCudaErrors(hipMemcpy(output + 3 * map_w * map_h * (globalCfg.startSeed / 5), dOutput + 3 * map_w * map_h * (globalCfg.startSeed / 5), 3 * map_w * map_h, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(output, dOutput, outputSize, hipMemcpyDeviceToHost));

	checkCudaErrors(hipFree(dTileData));
	checkCudaErrors(hipFree(dOutput));
	checkCudaErrors(hipFree(dMapData));
	checkCudaErrors(hipFree(dMiscMem));
	checkCudaErrors(hipFree(dVisitedMem));

	free(tileData);
	chrono::steady_clock::time_point time2 = chrono::steady_clock::now();
	std::chrono::nanoseconds duration = time2 - time1;
	printf("%i ms\n", (int)(duration.count() / 1000000));


	std::ofstream f = ofstream("output.bin", std::ios::binary);
	f.write((char*)output, outputSize);
	f.close();
	free(output);
}