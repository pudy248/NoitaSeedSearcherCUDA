#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include "misc/error.h"

#include "Precheckers.h"
#include "Worldgen.h"
#include "WorldgenSearch.h"
#include "Filters.h"

#include <iostream>
#include <fstream>
#include <chrono>
#include <thread>

//#define SEED_OUTPUT

//tired of seeing an error for it being undefined
__device__ int atomicAdd(int* address, int val);

struct GlobalConfig
{
	uint startSeed;
	uint endSeed;
	int printInterval;
	bool doWorldgen;
};

struct MemBlockSizes
{
	size_t outputSize;
	size_t mapDataSize;
	size_t miscMemSize;
	size_t visitedMemSize;
};

__global__ void Kernel(byte* outputBlock, byte* dMapData, byte* dMiscMem, byte* dVisitedMem, MemBlockSizes memSizes, GlobalConfig globalCfg, PrecheckConfig precheckCfg, WorldgenConfig worldCfg, LootConfig lootCfg, FilterConfig filterCfg, int* checkedSeeds, int* passedSeeds)
{
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	uint stride = blockDim.x * gridDim.x;

	for (int seed = globalCfg.startSeed + index; seed < globalCfg.endSeed; seed += stride)
	{
#ifdef SEED_OUTPUT
		byte* output = outputBlock + (seed - globalCfg.startSeed) * memSizes.outputSize;
#else
		byte* output = outputBlock + index * memSizes.outputSize;
#endif

		byte* map = dMapData + index * memSizes.mapDataSize;
		byte* miscMem = dMiscMem + index * memSizes.miscMemSize;
		byte* visited = dVisitedMem + index * memSizes.visitedMemSize;
		byte* spawnableMem = miscMem;

		if (!PrecheckSeed(seed, precheckCfg))
		{
			atomicAdd(checkedSeeds, 1);
			continue;
		}

		bool seedPassed = true;

		//if (globalCfg.doWorldgen)
		//{
		//	GenerateMap(seed, output, map, visited, miscMem, worldCfg, globalCfg.startSeed / 5);

		//	CheckSpawnables(map, seed, spawnableMem, output, worldCfg, lootCfg, memSizes.miscMemSize);
		
		//	SpawnableBlock result = ParseSpawnableBlock(spawnableMem, map, output, lootCfg, memSizes.mapDataSize);
		//	seedPassed = SpawnablesPassed(result, filterCfg, true);
		//}

		atomicAdd(checkedSeeds, 1);
		if (seedPassed)
			atomicAdd(passedSeeds, 1);
	}
}

/*
__global__ void wandExperiment(const int level, const bool nonShuffle)
{
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	uint stride = blockDim.x * gridDim.x;

	constexpr int radius = 100;
	constexpr int seed = 913380622;
	constexpr int center_x = 5061;
	constexpr int center_y = 11119;
	for (int x = -radius + index + center_x; x < radius + center_x; x += stride) {
		for (int y = -radius + center_y; y < radius + center_y; y++) {
			Wand w = GetWandWithLevel(seed, x, y, level, nonShuffle, false);
			bool found = false;
			for (int i = 0; i < w.spellIdx; i++)
			{
				if (w.spells[i] == SPELL_SWAPPER_PROJECTILE)
					found = true;
			}

			if (found) printf("%i %i\n", x, y);
		}
	}
}
*/

int main()
{
	/*
	hipSetDeviceFlags(hipDeviceMapHost);

	volatile int* h_buckets;
	volatile int* d_buckets;

	hipHostAlloc((void**)&h_buckets, sizeof(volatile int) * 70, hipHostMallocMapped);

	hipHostGetDevicePointer((void**)&d_buckets, (void*)h_buckets, 0);

	for (int i = 0; i < 70; i++) h_buckets[i] = 0;
	wandExperiment<<<256,64>>>((int*)d_buckets, 6, true);
	hipDeviceSynchronize();
	for (int i = 0; i < 70; i++) {
		printf("multicast %i: %i wands\n", i, h_buckets[i]);
	}
	return;*/

	for (int global_iters = 0; global_iters < 1; global_iters++)
	{
		chrono::steady_clock::time_point time1 = chrono::steady_clock::now();

		//MINES
		WorldgenConfig worldCfg = { 348, 448, 256, 103, 34, 14, true, false, 100 };
		const char* fileName = "minesWang.bin";
		constexpr auto NUMBLOCKS = 128;
		constexpr auto BLOCKSIZE = 64;
		constexpr auto biomeIdx = 0;
		constexpr auto mapMemMult = 4;
		constexpr auto miscMemMult = 10;

		//EXCAVATION SITE
		//WorldgenConfig worldCfg = { 344, 440, 409, 102, 31, 17, false, false, 100 };
		//const char* fileName = "excavationsiteWang.bin";
		//constexpr auto NUMBLOCKS = 64;
		//constexpr auto BLOCKSIZE = 64;
		//constexpr auto biomeIdx = 1;
		//constexpr auto mapMemMult = 4;
		//constexpr auto miscMemMult = 10;

		//SNOWCAVE
		//WorldgenConfig worldCfg = { 440, 560, 512, 153, 30, 20, false, false, 100 };
		//const char* fileName = "snowcaveWang.bin";
		//constexpr auto NUMBLOCKS = 64;
		//constexpr auto BLOCKSIZE = 32;
		//constexpr auto biomeIdx = 1;
		//constexpr auto mapMemMult = 4;
		//constexpr auto miscMemMult = 10;

		//CRYPT
		//WorldgenConfig worldCfg = { 282, 342, 717, 204, 26, 35, false, false, 100 };
		//const char* fileName = "cryptWang.bin";
		//constexpr auto NUMBLOCKS = 32;
		//constexpr auto BLOCKSIZE = 32;
		//constexpr auto biomeIdx = 10;
		//constexpr auto mapMemMult = 4;
		//constexpr auto miscMemMult = 10;

		//OVERGROWN CAVERNS
		//WorldgenConfig worldCfg = { 144, 235, 359, 461, 59, 16, false, false, 1 };
		//const char* fileName = "fungiforestWang.bin";
		//constexpr auto NUMBLOCKS = 32;
		//constexpr auto BLOCKSIZE = 32;
		//constexpr auto biomeIdx = 15;
		//constexpr auto mapMemMult = 4;
		//constexpr auto miscMemMult = 10;

		//HELL
		//WorldgenConfig worldCfg = { 156, 364, 921, 256, 25, 43, false, true, 100 };
		//const char* fileName = "hellWang.bin";
		//constexpr auto NUMBLOCKS = 32;
		//constexpr auto BLOCKSIZE = 32;
		//constexpr auto biomeIdx = 0;
		//constexpr auto mapMemMult = 4;
		//constexpr auto miscMemMult = 10;

		MemBlockSizes memSizes = {
#ifdef SEED_OUTPUT
			3 * worldCfg.map_w * worldCfg.map_h,
#else
			256,
#endif
			mapMemMult * 3 * worldCfg.map_w * (worldCfg.map_h + 4),
			miscMemMult * worldCfg.map_w * worldCfg.map_h,
			worldCfg.map_w * worldCfg.map_h
		};

		GlobalConfig globalCfg = { 1, INT_MAX, 5, true };

		Item iF1[FILTER_OR_COUNT] = { WAND_T1 };
		Item iF2[FILTER_OR_COUNT] = { MIMIC };
		Spell sF1[FILTER_OR_COUNT] = { SPELL_SPEED, SPELL_ACCELERATING_SHOT, SPELL_LIGHT_SHOT, SPELL_GRAVITY };
		Spell sF2[FILTER_OR_COUNT] = { SPELL_BLACK_HOLE_DEATH_TRIGGER, SPELL_BLACK_HOLE };
		//Spell sF3[FILTER_OR_COUNT] = { SPELL_BLACK_HOLE };

		ItemFilter iFilters[] = { ItemFilter(iF1), ItemFilter(iF2) };
		Material mFilters[] = { FUNGUS_POWDER };
		SpellFilter sFilters[] = { SpellFilter(sF1), SpellFilter(sF2) };

		FilterConfig filterCfg = FilterConfig(false, 0, iFilters, 0, mFilters, 0, sFilters, false, 1);
		LootConfig lootCfg = LootConfig(0, true, false, false, false, false, filterCfg.materialFilterCount > 0, true, biomeIdx, false);

		PrecheckConfig precheckCfg = {
			true,
			false, WATER,
			false, WATER,
			true, {MUD, WATER, SOIL}, {MUD, WATER, SOIL},
			false, true, {FungalShift(SS_ACID_GAS, false, SD_NONE, true)},
			false, {BM_GOLD_VEIN_SUPER, BM_NONE, BM_NONE},
			false, {{PERK_PERKS_LOTTERY, true, 0, 3}, {PERK_UNLIMITED_SPELLS, false, 0, 6}, {PERK_EDIT_WANDS_EVERYWHERE, false, 0, 3}, {PERK_PROTECTION_EXPLOSION, false, 0, 6}, {PERK_NO_MORE_SHUFFLE, false, 0, 6}},
			false, filterCfg, lootCfg
		};

		if (precheckCfg.checkBiomeModifiers && !ValidateBiomeModifierConfig(precheckCfg))
		{
			printf("Impossible biome modifier set! Aborting...\n");
			return;
		}

		int sharedMemSize = 0;
#ifdef SEED_OUTPUT
		size_t outputSize = (globalCfg.endSeed - globalCfg.startSeed) * memSizes.outputSize;
#else
		size_t outputSize = NUMBLOCKS * BLOCKSIZE * memSizes.outputSize;
#endif
		size_t tileDataSize = 3 * worldCfg.tiles_w * worldCfg.tiles_h;
		size_t mapDataSize = NUMBLOCKS * BLOCKSIZE * memSizes.mapDataSize;
		size_t miscMemSize = NUMBLOCKS * BLOCKSIZE * memSizes.miscMemSize;
		size_t visitedMemSize = NUMBLOCKS * BLOCKSIZE * memSizes.visitedMemSize;

		byte* output = (byte*)malloc(outputSize);
		byte* tileData = (byte*)malloc(3 * worldCfg.tiles_w * worldCfg.tiles_h);
		std::ifstream source(fileName, std::ios_base::binary);
		source.read((char*)tileData, 3 * worldCfg.tiles_w * worldCfg.tiles_h);
		source.close();

		byte* dTileData;
		byte* dOutput;
		byte* dMapData;
		byte* dMiscMem;
		byte* dVisitedMem;

		printf("Memory Usage Statistics:\n");
		printf("Output: %iMB  Map data: %iMB\n", outputSize / 1000000, mapDataSize / 1000000);
		printf("Misc memory: %iMB  Visited cells: %iMB\n", miscMemSize / 1000000, visitedMemSize / 1000000);
		printf("Total memory: %iMB\n",(tileDataSize + outputSize + mapDataSize + miscMemSize + visitedMemSize) / 1000000);

		hipSetDeviceFlags(hipDeviceMapHost);

		volatile int* h_checkedSeeds, * h_passedSeeds;
		volatile int* d_checkedSeeds, * d_passedSeeds;

		hipHostAlloc((void**)&h_checkedSeeds, sizeof(volatile int), hipHostMallocMapped);
		hipHostAlloc((void**)&h_passedSeeds, sizeof(volatile int), hipHostMallocMapped);

		hipHostGetDevicePointer((void**)&d_checkedSeeds, (void*)h_checkedSeeds, 0);
		hipHostGetDevicePointer((void**)&d_passedSeeds, (void*)h_passedSeeds, 0);

		*h_checkedSeeds = 0;
		*h_passedSeeds = 0;

		checkCudaErrors(hipMalloc(&dTileData, tileDataSize));
		checkCudaErrors(hipMalloc(&dOutput, outputSize));
		checkCudaErrors(hipMalloc(&dMapData, mapDataSize));
		checkCudaErrors(hipMalloc(&dMiscMem, miscMemSize));
		checkCudaErrors(hipMalloc(&dVisitedMem, visitedMemSize));

		checkCudaErrors(hipMemcpy(dTileData, tileData, 3 * worldCfg.tiles_w * worldCfg.tiles_h, hipMemcpyHostToDevice));
		buildTS << <1, 1 >> > (dTileData, worldCfg.tiles_w, worldCfg.tiles_h);
		checkCudaErrors(hipDeviceSynchronize());

		hipEvent_t _event;
		checkCudaErrors(hipEventCreateWithFlags(&_event, hipEventDisableTiming));
		Kernel << <NUMBLOCKS, BLOCKSIZE, sharedMemSize >> > (dOutput, dMapData, dMiscMem, dVisitedMem, memSizes, globalCfg, precheckCfg, worldCfg, lootCfg, filterCfg, (int*)d_checkedSeeds, (int*)d_passedSeeds);
		checkCudaErrors(hipEventRecord(_event));

		int intervals = 0;
		if (globalCfg.printInterval > 0)
		{
			while (hipEventQuery(_event) != hipSuccess && (*h_checkedSeeds) < (globalCfg.endSeed - globalCfg.startSeed))
			{
				intervals++;
				float percentComplete = ((float)(*h_checkedSeeds) / (globalCfg.endSeed - globalCfg.startSeed));
				printf("Interval %i: %2.4f%% complete (%i seeds), found %i valid seeds.\n", intervals, percentComplete * 100, *h_checkedSeeds, *h_passedSeeds);
				this_thread::sleep_for(chrono::seconds(globalCfg.printInterval));
			}
		}
		checkCudaErrors(hipDeviceSynchronize());

		printf("Intervals elapsed: %i. Checked %i seeds, found %i valid seeds.\n", intervals, *h_checkedSeeds, *h_passedSeeds);

		freeTS << <1, 1 >> > ();
		checkCudaErrors(hipDeviceSynchronize());

		checkCudaErrors(hipMemcpy(output, dOutput, outputSize, hipMemcpyDeviceToHost));

		checkCudaErrors(hipFree(dTileData));
		checkCudaErrors(hipFree(dOutput));
		checkCudaErrors(hipFree(dMapData));
		checkCudaErrors(hipFree(dMiscMem));
		checkCudaErrors(hipFree(dVisitedMem));

		free(tileData);
		chrono::steady_clock::time_point time2 = chrono::steady_clock::now();
		std::chrono::nanoseconds duration = time2 - time1;
		printf("%i ms\n", (int)(duration.count() / 1000000));


		std::ofstream f = ofstream("output.bin", std::ios::binary);
		f.write((char*)output, outputSize);
		f.close();
		free(output);
	}
}